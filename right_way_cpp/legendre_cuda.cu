#include "hip/hip_runtime.h"
#include <torch/torch.h>
#include <cstdio>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define DIVUP(m, n) ((m + n - 1) / n)
#define INDEX3D(a, b, c, db, dc) (((a) * (db) * (dc) + (b) * (dc) + (c)))

__global__ void leg_fwd_kernel(const torch::PackedTensorAccessor64<float, 2> x, 
                                torch::PackedTensorAccessor64<float, 3> leg, 
                                int batch_size, int in_feats, int degree, int numThreads){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numThreads) {
        int irow = idx / in_feats;
        int icol = idx % in_feats;
        
        // error location:
        float x_val = x[irow][icol];
        leg[1][irow][icol] = x_val;
        float leg_val_z = x_val; // index = i - 1
        float leg_val_zz = 1;    // index = i - 2

        for(int d = 2; d < degree + 1; d++){
            float df = static_cast<float>(d);
            float denom_inv = 1.f / df;
            float new_leg_val = ((2 * df - 1) * x_val * leg_val_z - (df - 1) * x_val * leg_val_zz) * denom_inv;
            leg[d][irow][icol] = new_leg_val;


            // finally
            leg_val_zz = leg_val_z;
            leg_val_z = new_leg_val;
        }
    }
}


__global__ void leg_bwd_kernel(const torch::PackedTensorAccessor64<float, 3> gout, 
                                const torch::PackedTensorAccessor64<float, 2> x, 
                                const torch::PackedTensorAccessor64<float, 3> leg, 
                                torch::PackedTensorAccessor64<float, 2> grad_x, 
                                int batch_size, int in_feats, int degree, int numThreads){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numThreads) {

        int irow = idx / in_feats;
        int icol = idx % in_feats;

        float b0 = 0, b1 = 1;
        float b_z = b1, b_zz = b0; // b(i-1) and b(i-2)

        float x_val = x[irow][icol];

        // grad wrt d=0 is equal to zero
        // here is grad wrt d=1
        float grad_x_val = gout[1][irow][icol];

        for(int d = 2; d < degree + 1; d++){

            float df = static_cast<float>(d);
            float denom_inv = 1.f / df;

            // 2a(i-1)
            float b = ((2 * df - 1) * (leg[d-1][irow][icol] + x_val * b_z) - (df - 1) * b_zz) * denom_inv;

            grad_x_val += gout[d][irow][icol] * b;

            // finally
            b_zz = b_z;
            b_z = b;
        }

        grad_x[irow][icol] = grad_x_val;


    }
}



void leg_launcher(const torch::PackedTensorAccessor64<float, 2> x, 
                    torch::PackedTensorAccessor64<float, 3> leg, 
                    int batch_size, int in_feats, int degree){

    int numThreads = batch_size * in_feats;
    dim3 blockSize(DIVUP(numThreads, THREADS_PER_BLOCK));
    dim3 threadSize(THREADS_PER_BLOCK);
    leg_fwd_kernel<<<blockSize, threadSize>>>(x, leg, batch_size, in_feats, degree, numThreads);
}

void leg_bwd_launcher(const torch::PackedTensorAccessor64<float, 3> gout, 
                    const torch::PackedTensorAccessor64<float, 2> x, 
                    const torch::PackedTensorAccessor64<float, 3> leg, 
                    torch::PackedTensorAccessor64<float, 2> grad_x, 
                    int batch_size, int in_feats, int degree){

    int numThreads = batch_size * in_feats;
    dim3 blockSize(DIVUP(numThreads, THREADS_PER_BLOCK));
    dim3 threadSize(THREADS_PER_BLOCK);
    leg_bwd_kernel<<<blockSize, threadSize>>>(gout, x, leg, grad_x, batch_size, in_feats, degree, numThreads);
}


